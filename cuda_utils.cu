#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"


namespace cuda_utils {

    // Call hipMemset on padded before inserting
    __global__ void pad_array(
        hipfftDoubleComplex* __restrict padded_array,
        const hipfftDoubleComplex* __restrict unpadded_array,
        unsigned int N_padded, unsigned int N
    ) {
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        if (tid < N * N) {
            unsigned int x = (tid & (N_padded - 1));
            unsigned int y = tid / N_padded;
            unsigned int x_unpadded, y_unpadded;

            unsigned int first_nonzero_index = (N_padded - N) / 2;

            // Type conversions necessary to allow for negative indices
            bool center_site = (first_nonzero_index <= x)
                && (x < first_nonzero_index + N)
                && (first_nonzero_index <= y)
                && (y < first_nonzero_index + N);

            if (center_site) {
                x_unpadded = x - first_nonzero_index;
                y_unpadded = y - first_nonzero_index;
                padded_array[y * N_padded + x] = unpadded_array[y_unpadded * N + x_unpadded];
            }
        }
    }

    __global__ void unpad_array(
        const hipfftDoubleComplex* __restrict padded_array,
        hipfftDoubleComplex* __restrict unpadded_array,
        unsigned int N_padded, unsigned int N
    ) {
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        unsigned int x = (tid & (N_padded - 1));
        unsigned int y = tid / N_padded;
        unsigned int x_unpadded, y_unpadded;

        unsigned int first_nonzero_index = (N_padded - N) / 2;


        bool center_site = (first_nonzero_index <= x)
                        && (x < first_nonzero_index + N)
                        && (first_nonzero_index <= y)
                        && (y < first_nonzero_index + N);
                            

        if (center_site) {
            x_unpadded = x - first_nonzero_index;
            y_unpadded = y - first_nonzero_index;
            unpadded_array[y_unpadded * N + x_unpadded] = padded_array[y * N_padded + x];
        }
    }

    // Shift the FFT output such that both axis go from
    // -|k_max_i| to k_max_i for i in x, y
    __global__ void fft_shift(
        hipfftDoubleComplex* arr_shifted,
        const hipfftDoubleComplex* arr_unshifted,
        unsigned int N_x, unsigned int N_y
    ) {
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        unsigned int x = tid % N_x;
        unsigned int y = tid / N_x;

        unsigned int x_new = (x + N_x / 2) % N_x;
        unsigned int y_new = (y + N_y / 2) % N_y;

        arr_shifted[y_new * N_x + x_new] = arr_unshifted[y * N_x + x];
    }

    //__global__ void fresnel_propagator_rad(
    //    hipfftDoubleComplex* dst,
    //    double delta_z_px,
    //    unsigned int number_of_pixels_padded,
    //    unsigned int number_of_pixels_unpadded,
    //    double focal_length_px, 
    //    double wavelength_px
    //) {
    //    const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //    
    //    const int x = tid % number_of_pixels_padded;
    //    const int y = tid / number_of_pixels_padded;
    //    
    //    const int x_rel = int(x - number_of_pixels_padded / 2);
    //    const int y_rel = int(y - number_of_pixels_padded / 2);


    //    if (math_utils::is_in_circle(x_rel, y_rel, 0, 0, number_of_pixels_unpadded / 2)) {

    //        // lens(-z) is not - lens(z) therefore the phase is substracted if delta_z < 0

    //        double sign = (delta_z_px < 0) ? -1.0: 1.0;
    //        delta_z_px = sign * delta_z_px;

    //        //double phase_rad =
    //        //    math_utils::PI() / wavelength_px * (1.0 / (focal_length_px + delta_z_px) - 1.0 / focal_length_px)
    //        //    * (pow(double(x_rel), 2.0) + pow(double(y_rel), 2.0));
    //            
    //        double phase_rad =
    //            math_utils::PI() / wavelength_px * (delta_z_px / pow(focal_length_px, 2.0))
    //            * (pow(double(x_rel), 2.0) + pow(double(y_rel), 2.0));

    //        // phase_rad = fmod(phase_rad, 2 * math_utils::PI());

    //        const double re = 1.0 / sqrt(2.0) * cos(sign * phase_rad);
    //        const double im = 1.0 / sqrt(2.0) * sin(sign * phase_rad);
    //                    
    //        const double re_prev = dst[tid].x;
    //        const double im_prev = dst[tid].y;

    //        dst[tid].x = (re * re_prev - im * im_prev);
    //        dst[tid].y = (re * im_prev + im * re_prev);
    //    }
    //}

    __global__ void multiply_by_quadratic_phase_factor(
        hipfftDoubleComplex* dst,
        unsigned int number_of_pixels_padded,
        double c
    ) {
        const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
            
        const int x = tid % number_of_pixels_padded;
        const int y = tid / number_of_pixels_padded;
            
        const auto x_rel = x - int(number_of_pixels_padded) / 2;
        const auto y_rel = y - int(number_of_pixels_padded) / 2;

        const double phase_rad = c * (x_rel * x_rel + y_rel * y_rel);


        const double re = 1.0 / sqrt(2.0) * cos(phase_rad);
        const double im = 1.0 / sqrt(2.0) * sin(phase_rad);
                            
        const double re_prev = dst[tid].x;
        const double im_prev = dst[tid].y;

        dst[tid].x = (re * re_prev - im * im_prev);
        dst[tid].y = (re * im_prev + im * re_prev);
    }

    __global__ void shifted_intensity_distribution(
        double* __restrict dst,
        const hipfftDoubleComplex* __restrict src,
        unsigned int N_x, unsigned int N_y
    ) {

        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        if (tid < N_x * N_y) {
            unsigned int x = tid % N_x;
            unsigned int y = tid / N_x;

            unsigned int x_new = (x + N_x / 2) % N_x;
            unsigned int y_new = (y + N_y / 2) % N_y;

            dst[y_new * N_x + x_new] = math_utils::intensity(src[y * N_x + x]);
        }
    }

    __global__ void set_phase_only_array(
        hipfftDoubleComplex* __restrict dst,
        const double* __restrict phasemap
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        dst[tid].x = 1.0 * cos(phasemap[tid]);
        dst[tid].y = 1.0 * sin(phasemap[tid]);
    }

    __global__ void extract_phasemap(
        double* __restrict dst,
        const hipfftDoubleComplex* __restrict src,
        bool overwrite
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (overwrite) {
            dst[tid] = math_utils::phase(src[tid]);
        }
        else {
            dst[tid] += math_utils::phase(src[tid]);
        }
    }

    __global__ void reset_fft_array(
        hipfftDoubleComplex* dst,
        double val
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        dst[tid].x = val;
        dst[tid].y = val;
    }

    __global__ void scale_array(
        double* dst,
        double scale_factor
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        dst[tid] *= scale_factor;
    }

    __global__ void scale_array(
        hipfftDoubleComplex* dst,
        double scale_factor
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        dst[tid].x *= scale_factor;
        dst[tid].y *= scale_factor;
    }

    __global__ void add_phase(
        hipfftDoubleComplex* __restrict dst,
        const double* __restrict src,
        unsigned int number_of_pixels_padded
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        const int x = tid % number_of_pixels_padded;
        const int y = tid / number_of_pixels_padded;

        const double phase_rad = src[tid];

        const double re = 1.0 / sqrt(2.0) * cos(phase_rad);
        const double im = 1.0 / sqrt(2.0) * sin(phase_rad);

        const double re_prev = dst[tid].x;
        const double im_prev = dst[tid].y;

        dst[tid].x = (re * re_prev - im * im_prev);
        dst[tid].y = (re * im_prev + im * re_prev);


    }

    double get_norm(
        const hipfftDoubleComplex* src,
        unsigned int size
    ) {
        double total = 0.0;
        std::for_each(src, src + size, [&total](const auto& c) {
            total += math_utils::intensity(c);
        });
        return total;
    }

    __global__ void simulate_two_FFTs_in_a_row(
        hipfftDoubleComplex* __restrict dst,
        const hipfftDoubleComplex* __restrict src,
        unsigned int width, unsigned int height
    ) {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

        const unsigned int x_old = tid % width;
        const unsigned int y_old = tid / width;
        // To flip an array an index i gets mapped to length - i - 1 because indices run from 0 to length - 1.
        // However, applying an FFT twice results in the array flipped AND shifted by 1, so -1 + 1 = 0
        const unsigned int x_new = math_utils::mod(width - x_old, width);
        const unsigned int y_new = math_utils::mod(height - y_old, height);

        dst[y_new * width + x_new] = src[y_old * width + x_old];
    }

    void save_phasemap(
        const std::string& filename,
        const hipfftDoubleComplex* arr,
        unsigned int width, unsigned int height
    ) {
        const auto extract_phase = [pi = math_utils::PI()](const hipfftDoubleComplex& c) {
            return byte(255.0 * math_utils::phase(c) / (2 * pi));
        };
        basic_fileIO::save_as_bmp<hipfftDoubleComplex>(filename, arr, width, height, extract_phase);
    }
}